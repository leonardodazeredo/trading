//nvcc vetor_runtime.cu -arch=sm_20  -o vetor
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <cmath>
#include <ctime>

#define DADOS_PATH "../DadosHistoricos/tabela_YHOO_bollinger.bin"

#define TAMANHO_PADRAO 10
#define DESVIOS_PADRAO 0.5

#define TAMANHO_MAXIMO 90
#define DESVIOS_MAXIMO 4

#define TAMANHO_INCREMENTO 1
#define DESVIOS_INCREMENTO 0.1

#define TAMANHO_INICIAL 2
#define DESVIOS_INICIAL 0.0

#define CAPITAL_INICIAL 0.0

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)


using namespace std;

__device__
float calcula_media(float* historico, int ini, int fim)
{
  int i;
  float sum = 0.0;
  for(i=ini; i<fim; i++)
  {
    sum+= historico[i];
  }

  return sum/(fim-ini);
}

__device__
float calcula_desvio(float* historico, int ini, int fim, float media)
{
  int i;
  float sum = 0.0;
  for(i=ini; i<fim; i++)
  {
    // sum+= pow((historico[i]-media),2.0);
     sum+= ((historico[i]-media)*(historico[i]-media));
  }

  sum = sum/(fim-ini);
  sum = sqrt(sum);

  return sum;
}

__global__  void 
trader_thread(float* historico, int* tamanhos_janelas, float* num_desvios, float* capital)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x; //???????????????????

   int tamanho = 0;

   int dia_corrente = tamanhos_janelas[i];
   int dia_anterior = tamanhos_janelas[i] - 1;
   float  media_movel;
   float desvio;

   while(dia_corrente<(tamanho + 1))
   {
      media_movel = calcula_media(historico, dia_corrente - tamanhos_janelas[i], dia_corrente);

      desvio = calcula_desvio(historico, dia_corrente - tamanhos_janelas[i], dia_corrente, media_movel);

      if(historico[dia_anterior] >= media_movel + (num_desvios[i])*desvio)
      {
         (capital[i])+= (historico[dia_corrente]);
      }
      else if(historico[dia_anterior] <= media_movel - (num_desvios[i])*desvio)
      {
         (capital[i])-= (historico[dia_corrente]);
      }

      dia_corrente++;
      dia_anterior++;
  }
}

int main (int argc, char **argv)
{
   FILE * pFile;

   int h_tamanho;

   int* h_tamanhos_janelas = NULL;
   int* d_tamanhos_janelas = NULL;

   float* h_close_prices_vet = NULL;
   float* h_num_desvios = NULL;
   float* h_capital = NULL;
   float* d_close_prices_vet = NULL;
   float* d_num_desvios = NULL;
   float* d_capital = NULL;

   int number_of_threads;

   if (argc != 1)
   {
      return EXIT_FAILURE;
   }
   else
   {
      pFile = fopen(DADOS_PATH , "rb" );

      if (pFile==NULL)
      {
         cout<<"Could not open file.\n" << "File error",stderr;
         return EXIT_FAILURE;
      }
      else
      {
         fread(&h_tamanho,sizeof(int),1,pFile);
         h_close_prices_vet = (float*) malloc(sizeof(float)*h_tamanho);

         fread(h_close_prices_vet,sizeof(float), h_tamanho,pFile);
         fclose (pFile);

         number_of_threads = ((TAMANHO_MAXIMO - TAMANHO_INICIAL)/TAMANHO_INCREMENTO + 1) * ((DESVIOS_MAXIMO - DESVIOS_INICIAL)/DESVIOS_INCREMENTO + 1);

         h_tamanhos_janelas = (int*) malloc(sizeof(int)*number_of_threads);
         h_num_desvios = (float*) malloc(sizeof(float)*number_of_threads);
         h_capital = (float*) malloc(sizeof(float)*number_of_threads);

         int i = 0;
         for (int w = TAMANHO_INICIAL; w <= (int) TAMANHO_MAXIMO ; w+=TAMANHO_INCREMENTO)
         {
           for (float d = DESVIOS_INICIAL; d <= (int) DESVIOS_MAXIMO; d+=DESVIOS_INCREMENTO)
           {
             h_tamanhos_janelas[i] = w;
             h_num_desvios[i] = d;
             h_capital[i] = CAPITAL_INICIAL;

             i++;
           }
         }

         float elapsedTimeGPU = 0.0f, elapsedTimeMEM = 0.0f;
         
         hipEvent_t e_Start,e_Stop;

         size_t free = 0, total = 0;
         cout << endl << "CUDA runtime versao: " << CUDART_VERSION << endl;
         
         //Reset no device
         CHECK_ERROR(hipDeviceReset());

         //Verificando espaço livre em memória
         CHECK_ERROR(hipMemGetInfo(&free, &total));
         cout << "Memoria livre: " << (free / 1024 / 1024)   << " MB\n";
         cout << "Memoria total: " << (total / 1024 / 1024)  << " MB\n";
         
         //Criando eventos
         CHECK_ERROR(hipEventCreate(&e_Start));
         CHECK_ERROR(hipEventCreate(&e_Stop));
         
         //Aloca memória GPU
         CHECK_ERROR(hipMalloc((void**) &d_close_prices_vet, h_tamanho * sizeof(float)));
         CHECK_ERROR(hipMalloc((void**) &d_tamanhos_janelas, number_of_threads * sizeof(int)));
         CHECK_ERROR(hipMalloc((void**) &d_num_desvios,      number_of_threads * sizeof(float)));
         CHECK_ERROR(hipMalloc((void**) &d_capital,          number_of_threads * sizeof(float)));
         
         //Copiando CPU --> GPU

         CHECK_ERROR(hipEventRecord(e_Start, hipEventDefault));
         
         CHECK_ERROR(hipMemcpy(d_close_prices_vet, h_close_prices_vet,          h_tamanho * sizeof(float),  hipMemcpyHostToDevice)); 
         CHECK_ERROR(hipMemcpy(d_tamanhos_janelas, h_tamanhos_janelas,  number_of_threads * sizeof(float),  hipMemcpyHostToDevice)); 
         CHECK_ERROR(hipMemcpy(d_num_desvios,      h_num_desvios,       number_of_threads * sizeof(float),  hipMemcpyHostToDevice)); 
         CHECK_ERROR(hipMemcpy(d_capital,          h_capital,           number_of_threads * sizeof(float),  hipMemcpyHostToDevice)); 
         
         CHECK_ERROR(hipEventRecord(e_Stop, hipEventDefault));
         CHECK_ERROR(hipEventSynchronize(e_Stop));
         CHECK_ERROR(hipEventElapsedTime(&elapsedTimeMEM, e_Start, e_Stop));
         CHECK_ERROR(hipEventRecord(e_Start, hipEventDefault));
         
         int numBlocks = 2;                                       //????????????????????????
         int threadsPerBlock = number_of_threads / numBlocks;     //????????????????????????
         
         
         trader_thread<<<numBlocks, threadsPerBlock>>> (d_close_prices_vet, d_tamanhos_janelas, d_num_desvios,d_capital);
         
         CHECK_ERROR(hipDeviceSynchronize());
         
         CHECK_ERROR(hipEventRecord(e_Stop, hipEventDefault));
         CHECK_ERROR(hipEventSynchronize(e_Stop));
         CHECK_ERROR(hipEventElapsedTime(&elapsedTimeGPU, e_Start, e_Stop));

         
         //Copiando GPU --> CPU
         float elapsedTime = 0.0f;
         CHECK_ERROR(hipEventRecord(e_Start, hipEventDefault));
         
         CHECK_ERROR(hipMemcpy(h_capital, d_capital, number_of_threads * sizeof(float),  hipMemcpyDeviceToHost));
         
         CHECK_ERROR(hipEventRecord(e_Stop, hipEventDefault));
         CHECK_ERROR(hipEventSynchronize(e_Stop));
         CHECK_ERROR(hipEventElapsedTime(&elapsedTime, e_Start, e_Stop));
         elapsedTimeMEM += elapsedTime;
        
         cout << endl << "Tempo gasto [MEM]: " << elapsedTimeMEM << " (ms)" << endl;
         cout << endl << "Tempo gasto [GPU]: " << elapsedTimeGPU << " (ms)" << endl;
         
         cout <<  endl << "Resultado: "<<  endl;


         CHECK_ERROR(hipFree(d_close_prices_vet));
         CHECK_ERROR(hipFree(d_tamanhos_janelas));
         CHECK_ERROR(hipFree(d_num_desvios));
         CHECK_ERROR(hipFree(d_capital));
         
         delete [] h_close_prices_vet;
         delete [] h_tamanhos_janelas;
         delete [] h_num_desvios;
         delete [] h_capital;

         // free(h_close_prices_vet);
         // free(h_tamanhos_janelas);
         // free(h_num_desvios);
         // free(h_capital);
         
         cout << "FIM" << endl;
         
         return EXIT_SUCCESS;
      }
   }
}
